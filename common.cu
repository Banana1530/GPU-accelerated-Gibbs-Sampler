#include "hip/hip_runtime.h"
#include "common.h"

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

int block_size(int thread_per_block, int total_thread){
    return (total_thread + thread_per_block - 1) / thread_per_block;
}

__global__ void trans_unif2exp(int n, float *u, float *thetaSq) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < n) {
        u[i] = (-1.0f/thetaSq[i]) * logf(u[i]);
    }
}

float *read_mat_col_maj(const char *a, int *row, int *col){
    std::ifstream myfile;
    myfile.open(a);

    myfile >> *row >> *col;
    float *mdata = (float *)malloc(sizeof(float) * (*row) * (*col));

    // Note: column major
    for (int i = 0; i < *row; i++) {       // i-th row
        for (int j = 0; j < *col; j++) {   // j-th col
            myfile >> mdata[j * (*row) + i];
        }
    }
    myfile.close();
    return mdata;
}

int print_mat_col_maj(float *a, int row, int col){
    for (int i = 0; i < row; i++) {       // i-th row
        for (int j = 0; j < col; j++) {   // j-th col
            std::cout <<  a[j * row + i] << "\t";
        }
        std::cout << std::endl;
    }
    return 0;
}

float *read_vec(const char *a, int *n){

    std::ifstream myfile;
    myfile.open(a);
    myfile >> *n;
    float *mdata = (float *)malloc(sizeof(float)* (*n));

    // Note: column major
    for (int i = 0; i < *n; i++) {       // i-th row
        myfile >> mdata[i];
    }
    myfile.close();
    return mdata;
}

int print_vec(float *a, int n){

    // Note: column major
    for (int i = 0; i < n; i++) {       // i-th row
        std::cout << a[i] << std::endl;
    }
    return 0;
}

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS         : return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED : return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED    : return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE   : return "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH   : return "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR   : return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR  : return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}

const char *curandGetErrorString(hiprandStatus_t error)
{
    switch (error)
    {
        case HIPRAND_STATUS_SUCCESS                  : return "HIPRAND_STATUS_SUCCESS";
        case HIPRAND_STATUS_VERSION_MISMATCH         : return "HIPRAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED          : return "HIPRAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED        : return "HIPRAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR               : return "HIPRAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE             : return "HIPRAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE      : return "HIPRAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED: return "HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED";
        case HIPRAND_STATUS_LAUNCH_FAILURE           : return "HIPRAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE      : return "HIPRAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED    : return "HIPRAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH            : return "HIPRAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR           : return "HIPRAND_STATUS_INTERNAL_ERROR";
    }
    return "<unknown>";
}

const char *cusolverGetErrorString(hipsolverStatus_t error)
{
    switch (error)
    {
        case HIPSOLVER_STATUS_SUCCESS                  : return "HIPSOLVER_STATUS_SUCCESS";
        case HIPSOLVER_STATUS_NOT_INITIALIZED          : return "HIPSOLVER_STATUS_NOT_INITIALIZED";
        case HIPSOLVER_STATUS_ALLOC_FAILED             : return "HIPSOLVER_STATUS_ALLOC_FAILED";
        case HIPSOLVER_STATUS_INVALID_VALUE            : return "HIPSOLVER_STATUS_INVALID_VALUE";
        case HIPSOLVER_STATUS_ARCH_MISMATCH            : return "HIPSOLVER_STATUS_ARCH_MISMATCH";
        case HIPSOLVER_STATUS_EXECUTION_FAILED         : return "HIPSOLVER_STATUS_EXECUTION_FAILED";
        case HIPSOLVER_STATUS_INTERNAL_ERROR           : return "HIPSOLVER_STATUS_INTERNAL_ERROR";
        case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }
    return "<unknown>";
}

int d_print_mat(const float *a,int n,int p){
    float *tmp = (float *)malloc(n*p*sizeof(float));
    CUDA_CALL(hipMemcpy(tmp,a,n*p*sizeof(float),hipMemcpyDeviceToHost));
    print_mat_col_maj(tmp,n,p);

    free(tmp);
    return 0;
}

__global__ void shrink_vector(float *d_vec, int n, float * d_scale){
    // TODO: Maybe put d_scale to shared mem?
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid < n){
        d_vec[tid] /= (*d_scale);
    }
}
